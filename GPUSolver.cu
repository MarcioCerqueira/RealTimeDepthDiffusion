#include "hip/hip_runtime.h"
#define VIENNACL_WITH_CUDA
#define PARALUTION 
#define CUSP
#define TILE_WIDTH 16

#include "GPUSolver.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/count.h>
#ifdef CUSP
	#include <cusp/ell_matrix.h>
	#include <cusp/csr_matrix.h>
	#include <cusp/monitor.h>
	#include <cusp/krylov/bicgstab.h>
	#include <cusp/relaxation/gauss_seidel.h>
	#include <cusp/relaxation/jacobi.h>
#endif
#ifdef PARALUTION
	#include <paralution.hpp>
#endif
#ifdef VIENNACL_WITH_CUDA
	#include <viennacl/compressed_matrix.hpp>
	#include <viennacl/ell_matrix.hpp>
	#include <viennacl/vector.hpp>
	#include <viennacl/linalg/bicgstab.hpp>
	#include <viennacl/linalg/jacobi_precond.hpp>
	#include <viennacl/linalg/ilu.hpp>
	#include <viennacl/linalg/amg.hpp>
#endif

//Global variables shared by the distinct solvers
float **devicePreviousImage;
float **deviceNextImage;
float **deviceError;
int **devicehorizontalIndexToWeight;
int **deviceverticalIndexToWeight;
__constant__ float deviceWeights[256];

__device__ float solveDiffusion(int left, int right, int up, int down, float *count, float sharedImage[][TILE_WIDTH + 2], int tidx, int tidy) {

	count[0] = 0;
	float weight = 0;
	float sum = 0;
	if (left < 256) {
		weight = deviceWeights[left];
		sum += weight * sharedImage[tidy][tidx - 1];
		count[0] += weight;
	}
	if (right < 256) {
		weight = deviceWeights[right];
		sum += weight * sharedImage[tidy][tidx + 1];
		count[0] += weight;
	}
	if (up < 256) {
		weight = deviceWeights[up];
		sum += weight * sharedImage[tidy - 1][tidx];
		count[0] += weight;
	}
	if (down < 256) {
		weight = deviceWeights[down];
		sum += weight * sharedImage[tidy + 1][tidx];
		count[0] += weight;
	}
	return sum / count[0];

}

void GPUCheckError(char *methodName) {

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess) printf("%s: %s\n", methodName, hipGetErrorString(error));
	
}

int divUp(int a, int b) { 
    return (a + b - 1)/b;
}

void GPUAllocateDeviceMemory(int rows, int cols, int levels) {
	
	devicePreviousImage = (float**)malloc(sizeof(float*) * levels);
	deviceNextImage = (float**)malloc(sizeof(float*) * levels);
	deviceError = (float**)malloc(sizeof(float*) * levels);
	devicehorizontalIndexToWeight = (int**)malloc(sizeof(int*) * levels);
	deviceverticalIndexToWeight = (int**)malloc(sizeof(int*) * levels);

	for(int level = 0; level < levels; level++) {
		int rowsPerLevel = rows / powf(2, level);
		int colsPerLevel = cols / powf(2, level);
		hipMalloc((void**)&devicePreviousImage[level], sizeof(float) * rowsPerLevel * colsPerLevel);
		hipMalloc((void**)&deviceNextImage[level], sizeof(float) * rowsPerLevel * colsPerLevel);
		hipMalloc((void**)&deviceError[level], sizeof(float) * rowsPerLevel * colsPerLevel);
		hipMalloc((void**)&devicehorizontalIndexToWeight[level], sizeof(int) * rowsPerLevel * colsPerLevel);
		hipMalloc((void**)&deviceverticalIndexToWeight[level], sizeof(int) * rowsPerLevel * colsPerLevel);
	}

#ifdef PARALUTION
	paralution::init_paralution();
#endif
	GPUCheckError("GPUAllocateDeviceMemory");
	
}

void GPUFreeDeviceMemory(int levels) {

	for(int level = 0; level < levels; level++) {
		hipFree(devicePreviousImage[level]);
		hipFree(deviceNextImage[level]);
		hipFree(deviceError[level]);
		hipFree(devicehorizontalIndexToWeight[level]);
		hipFree(deviceverticalIndexToWeight[level]);
	}

#ifdef PARALUTION
	paralution::stop_paralution();
#endif
	GPUCheckError("GPUFreeDeviceMemory");

}

__global__ void convert(unsigned char *src, size_t srcPitch, float *dst, size_t dstPitch, int rows, int cols)
{

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= cols || y >= rows) return;

	float *dstRow = (float*)((char*)dst + y * dstPitch);
	unsigned char *srcRow = src + y * srcPitch;
	dstRow[x] = srcRow[x * 3 + 0];

}

__global__ void pyrDown(unsigned char *prevScribbleImage, size_t prevScribblePitch, unsigned char *prevEditedImage, size_t prevEditedPitch, 
	int previousRows, int previousCols, unsigned char *currScribbleImage, size_t currScribblePitch, unsigned char *currEditedImage, 
	size_t currEditedPitch, int currentRows, int currentCols)
{
	
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= currentCols || y >= currentRows) return;
    int kernelSize = 2;
    for(int py = 2 * y - kernelSize/2; py < 2 * y + kernelSize/2; py++) {
		for(int px = 2 * x - kernelSize/2; px < 2 * x + kernelSize/2; px++) {
			int pixel = py * previousCols + px;
            if(pixel >= 0 && pixel < previousCols * previousRows) {
				unsigned char *prevScribbleImageRow = prevScribbleImage + py * prevScribblePitch;
				if(prevScribbleImageRow[px] == 255) {
					unsigned char *currScribbleImageRow = currScribbleImage + y * currScribblePitch;
					unsigned char *currEditedImageRow = currEditedImage + y * currEditedPitch;
					unsigned char *prevEditedImageRow = prevEditedImage + py * prevEditedPitch;
					currScribbleImageRow[x] = 255;
                    currEditedImageRow[x * 3 + 0] =  prevEditedImageRow[px * 3 + 0];
                }           
			}
        }
    }
	
}

__global__ void copyFromPinnedData(float *output, float *input, size_t inputPitch, int rows, int cols) 
{
	
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= cols || y >= rows) return;

	float *inputRow = (float*)((char*)input + y * inputPitch);
	int pixel = y * cols + x;
	output[pixel] = inputRow[x];

}

__global__ void copyToPinnedData(float *output, float *input, size_t outputPitch, int rows, int cols) 
{
	
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= cols || y >= rows) return;

	float *outputRow = (float*)((char*)output + y * outputPitch);
	int pixel = y * cols + x;
	outputRow[x] = input[pixel];

}

__global__ void loadIndexToWeight(unsigned char *grayImage, int *horizontalIndexToWeight, int *verticalIndexToWeight, size_t grayPitch, int rows, int cols)
{

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= cols || y >= rows) return;
	
	unsigned char *grayImageRow = grayImage + y * grayPitch;
	unsigned char *grayImageRowMinus = grayImage + (y - 1) * grayPitch;
	unsigned char *grayImageRowPlus = grayImage + (y + 1) * grayPitch;
	int tidx = threadIdx.x + 1;
	int tidy = threadIdx.y + 1;
	
	__shared__ unsigned char sharedGrayImage[TILE_WIDTH + 2][TILE_WIDTH + 2];
	sharedGrayImage[tidy][tidx] = grayImageRow[x];
	if(tidx == 1) sharedGrayImage[tidy][0] = grayImageRow[x - 1];
	if(tidx == TILE_WIDTH) sharedGrayImage[tidy][TILE_WIDTH + 1] = grayImageRow[x + 1];
	if(tidy == 1) sharedGrayImage[0][tidx] = grayImageRowMinus[x];
	if(tidy == TILE_WIDTH) sharedGrayImage[TILE_WIDTH + 1][tidx] = grayImageRowPlus[x];
	__syncthreads();
	
	unsigned char grayIntensity = sharedGrayImage[tidy][tidx];
	int directions[4] = {257, 257, 257, 257};
	if(x - 1 >= 0) directions[0] = abs(grayIntensity - sharedGrayImage[tidy][tidx - 1]);
	if(x + 1 < cols) directions[1] = abs(grayIntensity - sharedGrayImage[tidy][tidx + 1]);
	if(y - 1 >= 0) directions[2] = abs(grayIntensity - sharedGrayImage[tidy - 1][tidx]);
	if(y + 1 < rows) directions[3] = abs(grayIntensity - sharedGrayImage[tidy + 1][tidx]);
	horizontalIndexToWeight[y * cols + x] = directions[0] * 1000 + directions[1];
	verticalIndexToWeight[y * cols + x] = directions[2] * 1000 + directions[3];
}

__global__ void jacobi(float *output, float *input, int *horizontalIndexToWeight, int *verticalIndexToWeight, unsigned char *scribbleImage, float *error, size_t inputPitch, 
	size_t scribblePitch, int rows, int cols)
{

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
	
	if(x >= cols || y >= rows) return;

	int pixel = y * cols + x;
	int tidx = threadIdx.x + 1;
	int tidy = threadIdx.y + 1;
	
	__shared__ float sharedImage[TILE_WIDTH + 2][TILE_WIDTH + 2];
	float *inputRow = (float*)((char*)input + y * inputPitch);
	float *inputRowMinus = (float*)((char*)input + (y - 1) * inputPitch);
	float *inputRowPlus = (float*)((char*)input + (y + 1) * inputPitch);
	sharedImage[tidy][tidx] = inputRow[x];
	if(tidx == 1) sharedImage[tidy][0] = inputRow[x - 1];
	if(tidx == TILE_WIDTH) sharedImage[tidy][TILE_WIDTH + 1] = inputRow[x + 1];
	if(tidy == 1) sharedImage[0][tidx] = inputRowMinus[x];
	if(tidy == TILE_WIDTH) sharedImage[TILE_WIDTH + 1][tidx] = inputRowPlus[x];
	__syncthreads();
	
	unsigned char *scribbleImageRow = scribbleImage + y * scribblePitch;
	if(scribbleImageRow[x] == 255) return;
	            
	int index = horizontalIndexToWeight[pixel];
	int left = index / 1000;
	int right = index % 1000;
	index = verticalIndexToWeight[pixel];
	int up = index / 1000;
	int down = index % 1000;
	float count[1];
	float result = solveDiffusion(left, right, up, down, count, sharedImage, tidx, tidy);

	if(count > 0) {
		output[pixel] = result;
		error[pixel] = abs(result - sharedImage[tidy][tidx]);
	} else {
		error[pixel] = 0;
	}

}

__global__ void chebyshevSemiIterativeMethod(float *previousImage, float *output, float *input, int *horizontalIndexToWeight, int *verticalIndexToWeight, 
	unsigned char *scribbleImage, float *error, size_t inputPitch, size_t scribblePitch, float omega, float gamma, int rows, int cols)
{

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= cols || y >= rows) return;

	int pixel = y * cols + x;
	int tidx = threadIdx.x + 1;
	int tidy = threadIdx.y + 1;
	
	__shared__ float sharedImage[TILE_WIDTH + 2][TILE_WIDTH + 2];
	float *inputRow = (float*)((char*)input + y * inputPitch);
	float *inputRowMinus = (float*)((char*)input + (y - 1) * inputPitch);
	float *inputRowPlus = (float*)((char*)input + (y + 1) * inputPitch);
	sharedImage[tidy][tidx] = inputRow[x];
	if(tidx == 1) sharedImage[tidy][0] = inputRow[x - 1];
	if(tidx == TILE_WIDTH) sharedImage[tidy][TILE_WIDTH + 1] = inputRow[x + 1];
	if(tidy == 1) sharedImage[0][tidx] = inputRowMinus[x];
	if(tidy == TILE_WIDTH) sharedImage[TILE_WIDTH + 1][tidx] = inputRowPlus[x];
	__syncthreads();
	
	unsigned char *scribbleImageRow = scribbleImage + y * scribblePitch;
	if(scribbleImageRow[x] == 255) return;
	            
    int index = horizontalIndexToWeight[pixel];
	int left = index / 1000;
	int right = index % 1000;
	index = verticalIndexToWeight[pixel];
	int up = index / 1000;
	int down = index % 1000;
	float count[1];
	float result = solveDiffusion(left, right, up, down, count, sharedImage, tidx, tidy);

	if(count[0] > 0) {
		float previousColor = previousImage[pixel];
		output[pixel] = (omega * (gamma * (result - sharedImage[tidy][tidx]) + sharedImage[tidy][tidx] - previousColor)) + previousColor;
		previousImage[pixel] = sharedImage[tidy][tidx];
		error[pixel] = abs(result - sharedImage[tidy][tidx]);
	} else {
		error[pixel] = 0;
	}

}

__global__ void gaussSeidel(float *image, int *horizontalIndexToWeight, int *verticalIndexToWeight, unsigned char *scribbleImage, float *error, size_t imagePitch,
	size_t scribblePitch, int color, float omega, int rows, int cols)
{

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= cols || y >= rows) return;

	int pixel = y * cols + x;
	int tidx = threadIdx.x + 1;
	int tidy = threadIdx.y + 1;
	
	__shared__ float sharedImage[TILE_WIDTH + 2][TILE_WIDTH + 2];
	float *imageRow = (float*)((char*)image + y * imagePitch);
	float *imageRowMinus = (float*)((char*)image + (y - 1) * imagePitch);
	float *imageRowPlus = (float*)((char*)image + (y + 1) * imagePitch);
	sharedImage[tidy][tidx] = imageRow[x];
	if(tidx == 1) sharedImage[tidy][0] = imageRow[x - 1];
	if(tidx == TILE_WIDTH) sharedImage[tidy][TILE_WIDTH + 1] = imageRow[x + 1];
	if(tidy == 1) sharedImage[0][tidx] = imageRowMinus[x];
	if(tidy == TILE_WIDTH) sharedImage[TILE_WIDTH + 1][tidx] = imageRowPlus[x];
	__syncthreads();
	
	unsigned char *scribbleImageRow = scribbleImage + y * scribblePitch;
	if(scribbleImageRow[x] == 255) return;
	if(abs((x % 2) - (y % 2)) != color) return;
	            
    int index = horizontalIndexToWeight[pixel];
	int left = index / 1000;
	int right = index % 1000;
	index = verticalIndexToWeight[pixel];
	int up = index / 1000;
	int down = index % 1000;

	float count[1];
	float result = solveDiffusion(left, right, up, down, count, sharedImage, tidx, tidy);

	if(count[0] > 0) {
		error[pixel] = abs(result - sharedImage[tidy][tidx]);
		float depth = sharedImage[tidy][tidx];
		imageRow[x] = omega * (result - depth) + depth;
	} else {
		error[pixel] = 0;
	}

}

void GPULoadWeights(float beta) {
	
	float weights[256];
	for(int w = 0; w < 256; w++) weights[w] = expf(-beta * w);
	hipMemcpyToSymbol(HIP_SYMBOL(deviceWeights), weights, 256 * sizeof(float), 0, hipMemcpyHostToDevice);
	GPUCheckError("GPULoadWeights");

}

void GPUConvertToFloat(unsigned char *src, size_t srcPitch, float *dst, size_t dstPitch, int rows, int cols)
{
	dim3 threads(TILE_WIDTH, TILE_WIDTH);
    dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
	convert<<<grid, threads>>>(src, srcPitch, dst, dstPitch, rows, cols);
}

void GPUPyrDownAnnotation(unsigned char *prevScribbleImage, size_t prevScribblePitch, unsigned char *prevEditedImage, size_t prevEditedPitch, 
	int previousRows, int previousCols, unsigned char *currScribbleImage, size_t currScribblePitch, unsigned char *currEditedImage, 
	size_t currEditedPitch, int currentRows, int currentCols)
{

	dim3 threads(TILE_WIDTH, TILE_WIDTH);
    dim3 grid(divUp(currentCols, threads.x), divUp(currentRows, threads.y));
	pyrDown<<<grid, threads>>>(prevScribbleImage, prevScribblePitch, prevEditedImage, prevEditedPitch, previousRows, previousCols, 
		currScribbleImage, currScribblePitch, currEditedImage, currEditedPitch, currentRows, currentCols);

}

void GPUJacobi(float *depthImage, size_t depthPitch, unsigned char *scribbleImage, size_t scribblePitch, unsigned char *grayImage, 
	size_t grayPitch, int rows, int cols, float beta, int maxIterations, float tolerance, bool isDebugEnabled, bool chebyshevVariant, 
	int level)
{

	int iteration;
	float error;
	//Chebyshev's variant
	int S = 10;
	double omega;
	double rho = 0.99;
	double gamma = 0.99;
	
	dim3 threads(TILE_WIDTH, TILE_WIDTH);
    dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
	
	hipMemset(devicePreviousImage[level], 0, rows * cols * sizeof(float));
	copyFromPinnedData<<<grid, threads>>>(deviceNextImage[level], depthImage, depthPitch, rows, cols);
	loadIndexToWeight<<<grid, threads>>>(grayImage, devicehorizontalIndexToWeight[level], deviceverticalIndexToWeight[level], grayPitch, rows, cols);
	
	for(iteration = 0; iteration < maxIterations; iteration++) {
		
		if(!chebyshevVariant) {
		
			jacobi<<<grid, threads>>>(deviceNextImage[level], depthImage, devicehorizontalIndexToWeight[level], deviceverticalIndexToWeight[level], 
				scribbleImage, deviceError[level], depthPitch, scribblePitch, rows, cols);
		
		} else {
			
			if (iteration < S) omega = 1;
			else if (iteration == S) omega = 2.0 / (2.0 - rho * rho);
			else omega = 4.0 / (4.0 - rho * rho * omega);

			chebyshevSemiIterativeMethod<<<grid, threads>>>(devicePreviousImage[level], deviceNextImage[level], depthImage, 
				devicehorizontalIndexToWeight[level], deviceverticalIndexToWeight[level], scribbleImage, deviceError[level], depthPitch, scribblePitch, omega, gamma, rows, cols);

		} 
		
		copyToPinnedData<<<grid, threads>>>(depthImage, deviceNextImage[level], depthPitch, rows, cols);
    
		if(iteration % 100 == 0) {
			thrust::device_ptr<float> tptr = thrust::device_pointer_cast(deviceError[level]);
			error = thrust::reduce(tptr, tptr + rows * cols)/(rows * cols);
			if(error < tolerance) break;
		}
		
	}

	if (isDebugEnabled) std::cout << "Iterations: " << iteration << " | Error: " << error << std::endl;

}

void GPUGaussSeidel(float *depthImage, size_t depthPitch, unsigned char *scribbleImage, size_t scribblePitch, unsigned char *grayImage, 
	size_t grayPitch, int rows, int cols, float beta, int maxIterations, float tolerance, bool isDebugEnabled, int level)
{

	int iteration;
	float error;

	float omega = 1.9;
	
	dim3 threads(16, 16);
    dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
	int maxColors = 2;

	loadIndexToWeight<<<grid, threads>>>(grayImage, devicehorizontalIndexToWeight[level], deviceverticalIndexToWeight[level], grayPitch, rows, cols);
	for(iteration = 0; iteration < maxIterations; iteration++) {

		for(int color = 0; color < maxColors; color++)
			gaussSeidel<<<grid, threads>>>(depthImage, devicehorizontalIndexToWeight[level], deviceverticalIndexToWeight[level], scribbleImage, deviceError[level], depthPitch,
				scribblePitch, color, omega, rows, cols);
	
		if(iteration % 100 == 0) {
			thrust::device_ptr<float> tptr = thrust::device_pointer_cast(deviceError[level]);
			error = thrust::reduce(tptr, tptr + rows * cols)/(rows * cols);
			if(error < tolerance) break;
		}

	}
	if (isDebugEnabled) std::cout << "Iterations: " << iteration << " | Error: " << error << std::endl;

}

void CUSPJacobi(unsigned char *depthImage, unsigned char *scribbleImage, unsigned char *grayImage, int rows, int cols, 
	float beta, int maxIterations, float tolerance, bool isDebugEnabled)
{
#ifdef CUSP
    cusp::csr_matrix<int, float, cusp::host_memory> A(rows * cols, rows * cols, rows * cols * 5);
	cusp::array1d<float, cusp::host_memory> x(A.num_rows);
	cusp::array1d<float, cusp::host_memory> b(A.num_rows);
	
	for (int pixel = 0; pixel < rows * cols; pixel++) {
		x[pixel] = depthImage[pixel];
		b[pixel] = 0;
	}

	float weight;
	int counter = 0;
	for (int y = 0; y < rows; y++) {
		for (int x = 0; x < cols; x++) {
		
			int pixel = y * cols + x;
			A.row_offsets[pixel] = counter;

			if (scribbleImage[pixel] == 255) {
				A.column_indices[counter] = pixel;
				A.values[counter] = 1;
				b[pixel] = depthImage[pixel];
				counter++;
				continue;
			}
			
			float sum = 0;
			if (y > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y - 1) * cols + x]));
				A.column_indices[counter] = (y - 1) * cols + x;
				A.values[counter] = -weight;
				counter++;
				sum += weight;
			}
			if (x > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x - 1]));
				A.column_indices[counter] = y * cols + x - 1;
				A.values[counter] = -weight;
				counter++;
				sum += weight;
			}
			if (x < cols - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x + 1]));
				A.column_indices[counter] = y * cols + x + 1;
				A.values[counter] = -weight;
				sum += weight;
				counter++;
			}
			if (y < rows - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y + 1) * cols + x]));
				A.column_indices[counter] = (y + 1) * cols + x;
				A.values[counter] = -weight;
				counter++;
				sum += weight;
			}
			
			A.column_indices[counter] = pixel;
			A.values[counter] = sum;
			counter++;

		}
	}
	A.row_offsets[rows * cols] = counter;

	cusp::array1d<float, cusp::device_memory> d_x(x);
	cusp::array1d<float, cusp::device_memory> d_b(b);
	
	cusp::csr_matrix<int, float, cusp::device_memory> d_A(A);
	cusp::relaxation::jacobi<float, cusp::device_memory> M(d_A);
	cusp::array1d<float, cusp::device_memory> d_r(A.num_rows);
	cusp::multiply(d_A, d_x, d_r);
	cusp::blas::axpy(d_b, d_r, float(-1));
	
	cusp::monitor<float> monitor(d_b, maxIterations, tolerance, tolerance, false);
	
	while (!monitor.finished(d_r))
	{
		M(d_A, d_b, d_x);
		cusp::multiply(d_A, d_x, d_r);
		cusp::blas::axpy(d_b, d_r, float(-1));
		++monitor;
	}
	
	if(isDebugEnabled) monitor.print();
	cusp::array1d<float, cusp::host_memory> r(d_x);
	for (int pixel = 0; pixel < rows * cols; pixel++)
		depthImage[pixel] = r[pixel];
#else
	std::cout << "CUSP not supported" << std::endl;
#endif
}

void CUSPGaussSeidel(unsigned char *depthImage, unsigned char *scribbleImage, unsigned char *grayImage, int rows, int cols, 
	float beta, int maxIterations, float tolerance, bool isDebugEnabled)
{
#ifdef CUSP
    cusp::csr_matrix<int, float, cusp::host_memory> A(rows * cols, rows * cols, rows * cols * 5);
	cusp::array1d<float, cusp::host_memory> x(A.num_rows);
	cusp::array1d<float, cusp::host_memory> b(A.num_rows);
	
	for (int pixel = 0; pixel < rows * cols; pixel++) {
		x[pixel] = depthImage[pixel];
		b[pixel] = 0;
	}

	float weight;
	int counter = 0;
	for (int y = 0; y < rows; y++) {
		for (int x = 0; x < cols; x++) {
		
			int pixel = y * cols + x;
			A.row_offsets[pixel] = counter;

			if (scribbleImage[pixel] == 255) {
				A.column_indices[counter] = pixel;
				A.values[counter] = 1;
				b[pixel] = depthImage[pixel];
				counter++;
				continue;
			}
			
			float sum = 0;
			if (y > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y - 1) * cols + x]));
				A.column_indices[counter] = (y - 1) * cols + x;
				A.values[counter] = -weight;
				counter++;
				sum += weight;
			}
			if (x > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x - 1]));
				A.column_indices[counter] = y * cols + x - 1;
				A.values[counter] = -weight;
				counter++;
				sum += weight;
			}
			if (x < cols - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x + 1]));
				A.column_indices[counter] = y * cols + x + 1;
				A.values[counter] = -weight;
				sum += weight;
				counter++;
			}
			if (y < rows - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y + 1) * cols + x]));
				A.column_indices[counter] = (y + 1) * cols + x;
				A.values[counter] = -weight;
				counter++;
				sum += weight;
			}
			
			A.column_indices[counter] = pixel;
			A.values[counter] = sum;
			counter++;

		}
	}
	A.row_offsets[rows * cols] = counter;

	cusp::array1d<float, cusp::device_memory> d_x(x);
	cusp::array1d<float, cusp::device_memory> d_b(b);
	
	cusp::csr_matrix<int, float, cusp::device_memory> d_A(A);
	cusp::relaxation::gauss_seidel<float, cusp::device_memory> M(d_A);
	cusp::array1d<float, cusp::device_memory> d_r(A.num_rows);
	cusp::multiply(d_A, d_x, d_r);
	cusp::blas::axpy(d_b, d_r, float(-1));
	
	cusp::monitor<float> monitor(d_b, maxIterations, tolerance, tolerance, false);
	
	while (!monitor.finished(d_r))
	   {
		   M(d_A, d_b, d_x);
		   cusp::multiply(d_A, d_x, d_r);
		   cusp::blas::axpy(d_b, d_r, float(-1));
		   ++monitor;
	   }
	
	if(isDebugEnabled) monitor.print();
	cusp::array1d<float, cusp::host_memory> r(d_x);
	for (int pixel = 0; pixel < rows * cols; pixel++)
		depthImage[pixel] = r[pixel];
#else
	std::cout << "CUSP not supported" << std::endl;
#endif
}

void CUSPPCG(unsigned char *depthImage, unsigned char *scribbleImage, unsigned char *grayImage, int rows, int cols, float beta, 
	int maxIterations, float tolerance, bool isDebugEnabled)
{

#ifdef CUSP
	cusp::ell_matrix<int, float, cusp::host_memory> A(rows * cols, rows * cols, rows * cols * 5, 5);
	const int X = cusp::ell_matrix<int,float,cusp::host_memory>::invalid_index;
	cusp::array1d<float, cusp::host_memory> x(A.num_rows);
	cusp::array1d<float, cusp::host_memory> b(A.num_rows);
	
	for (int pixel = 0; pixel < rows * cols; pixel++) {
		x[pixel] = depthImage[pixel];
		b[pixel] = 0;
	}

	for (int y = 0; y < rows; y++) {
		for (int x = 0; x < cols; x++) {
			int pixel = y * cols + x;
			for(int w = 0; w < 5; w++) {
				A.column_indices(pixel, w) = X;
				A.values(pixel, w) = 0;
			}
		}
	}

	float weight;
	for (int y = 0; y < rows; y++) {
		for (int x = 0; x < cols; x++) {
		
			int counter = 0;
			int pixel = y * cols + x;
			if (scribbleImage[pixel] == 255) {
				A.column_indices(pixel, 0) = pixel;
				A.values(pixel, 0) = 1;
				b[pixel] = depthImage[pixel];
				continue;
			}
			
			float sum = 0;
			if (y > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y - 1) * cols + x]));
				A.column_indices(pixel, counter) = (y - 1) * cols + x;
				A.values(pixel, counter) = -weight;
				counter++;
				sum += weight;
			}
			if (x > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x - 1]));
				A.column_indices(pixel, counter) = y * cols + x - 1;
				A.values(pixel, counter) = -weight;
				counter++;
				sum += weight;
			}
			if (x < cols - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x + 1]));
				A.column_indices(pixel, counter) = y * cols + x + 1;
				A.values(pixel, counter) = -weight;
				sum += weight;
				counter++;
			}
			if (y < rows - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y + 1) * cols + x]));
				A.column_indices(pixel, counter) = (y + 1) * cols + x;
				A.values(pixel, counter) = -weight;
				counter++;
				sum += weight;
			}
			
			A.column_indices(pixel, counter) = pixel;
			A.values(pixel, counter) = sum;

		}
	}

	cusp::ell_matrix<int, float, cusp::device_memory> d_A(A);
	cusp::array1d<float, cusp::device_memory> d_x(x);
	cusp::array1d<float, cusp::device_memory> d_b(b);
	cusp::identity_operator<float, cusp::device_memory> M(A.num_rows, A.num_rows);
	cusp::monitor<float> monitor(d_b, maxIterations, tolerance, tolerance, false);
	cusp::krylov::bicgstab(d_A, d_x, d_b, monitor, M);
	
	if(isDebugEnabled) monitor.print();
	cusp::array1d<float, cusp::host_memory> r(d_x);
	for (int pixel = 0; pixel < rows * cols; pixel++)
		depthImage[pixel] = r[pixel];
#else
	std::cout << "CUSP not supported" << std::endl;
#endif
}

void ParalutionPCG(unsigned char *depthImage, unsigned char *scribbleImage, unsigned char *grayImage, int rows, int cols, float beta, 
	int maxIterations, float tolerance)
{

#ifdef PARALUTION
	float *h_x = (float*)malloc(rows * cols * sizeof(float));
	float *h_b = (float*)malloc(rows * cols * sizeof(float));
	int *row_offsets = (int*)malloc((rows * cols + 1) * sizeof(int));
	int *col = (int*)malloc(rows * cols * 5 * sizeof(int));
	float *val = (float*)malloc(rows * cols * 5 * sizeof(float));

	for (int pixel = 0; pixel < rows * cols; pixel++) {
		h_x[pixel] = depthImage[pixel];
		h_b[pixel] = 0;
	}

	float weight;
	int counter = 0;
	for (int y = 0; y < rows; y++) {
		for (int x = 0; x < cols; x++) {
		
			int pixel = y * cols + x;
			row_offsets[pixel] = counter;

			if (scribbleImage[pixel] == 255) {
				col[counter] = pixel;
				val[counter] = 1;
				h_b[pixel] = depthImage[pixel];
				counter++;
				continue;
			}
			
			float sum = 0;
			if (y > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y - 1) * cols + x]));
				col[counter] = (y - 1) * cols + x;
				val[counter] = -weight;
				counter++;
				sum += weight;
			}
			if (x > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x - 1]));
				col[counter] = y * cols + x - 1;
				val[counter] = -weight;
				counter++;
				sum += weight;
			}
			if (x < cols - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x + 1]));
				col[counter] = y * cols + x + 1;
				val[counter] = -weight;
				counter++;
				sum += weight;
			}
			if (y < rows - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y + 1) * cols + x]));
				col[counter] = (y + 1) * cols + x;
				val[counter] = -weight;
				counter++;
				sum += weight;
			}
			
			col[counter] = pixel;
			val[counter] = sum;
			counter++;

		}
	}
	row_offsets[rows * cols] = counter;

	paralution::LocalMatrix<float> A;
	paralution::LocalVector<float> x;
	paralution::LocalVector<float> b;

	A.AllocateCSR("A", rows * cols * 5, rows * cols, rows * cols);
	A.CopyFromCSR(row_offsets, col, val);
	A.ConvertToELL();
	x.Allocate("x", A.get_nrow());
	x.SetDataPtr(&h_x, "vector", A.get_nrow());
	b.Allocate("b", A.get_nrow());
	b.SetDataPtr(&h_b, "vector", A.get_nrow());
	
	paralution::BiCGStab<paralution::LocalMatrix<float>, paralution::LocalVector<float>, float > ls;
	paralution::Jacobi<paralution::LocalMatrix<float>, paralution::LocalVector<float>, float > p;
	
	A.MoveToAccelerator();
	x.MoveToAccelerator();
	b.MoveToAccelerator();
	ls.MoveToAccelerator();
	
	ls.Init(tolerance, tolerance, 1e+10, maxIterations);
	ls.SetOperator(A);
	ls.SetPreconditioner(p);
	ls.Verbose(0);
	ls.Build();
	ls.Solve(b, &x);
	
	x.MoveToHost();
	x.LeaveDataPtr(&h_x);

	for (int pixel = 0; pixel < rows * cols; pixel++)
		depthImage[pixel] = h_x[pixel];

	delete [] h_x;
	delete [] h_b;
	delete [] row_offsets;
	delete [] col;
	delete [] val;

	A.Clear();
	x.Clear();
	b.Clear();
	ls.Clear();
#else
	std::cout << "Paralution is not supported" << std::endl;
#endif

}

void ViennaCLPCG(unsigned char *depthImage, unsigned char *scribbleImage, unsigned char *grayImage, int rows, int cols, float beta, 
	int maxIterations, float tolerance)
{

#ifdef VIENNACL_WITH_CUDA
	std::vector<float> h_x(rows * cols);
	std::vector<float> h_b(rows * cols);
	std::vector< std::map< unsigned int, float> > h_A(rows * cols);

	for (int pixel = 0; pixel < rows * cols; pixel++) {
		h_x[pixel] = depthImage[pixel];
		h_b[pixel] = 0;
	}

	float weight;
	for (int y = 0; y < rows; y++) {
		for (int x = 0; x < cols; x++) {
		
			int pixel = y * cols + x;
			
			if (scribbleImage[pixel] == 255) {
				h_A[pixel][pixel] = 1;
				h_b[pixel] = depthImage[pixel];
				continue;
			}
			
			float sum = 0;
			if (y > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y - 1) * cols + x]));
				h_A[pixel][(y - 1) * cols + x] = -weight;
				sum += weight;
			}
			if (x > 0) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x - 1]));
				h_A[pixel][y * cols + x - 1] = -weight;
				sum += weight;
			}
			if (x < cols - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[y * cols + x + 1]));
				h_A[pixel][y * cols + x + 1] = -weight;
				sum += weight;
			}
			if (y < rows - 1) {
				weight = expf(-beta * fabs(grayImage[pixel] - grayImage[(y + 1) * cols + x]));
				h_A[pixel][(y + 1) * cols + x] = -weight;
				sum += weight;
			}
			
			h_A[pixel][pixel] = sum;

		}
	}

	viennacl::context ctx;
	viennacl::vector<float> x(rows * cols, ctx);
	viennacl::vector<float> b(rows * cols, ctx);
	viennacl::ell_matrix<float> A;
	viennacl::copy(h_x, x);
	viennacl::copy(h_b, b);
	viennacl::copy(h_A, A);
	
	viennacl::linalg::bicgstab_tag config(tolerance, maxIterations);
	config.abs_tolerance(tolerance);
	viennacl::linalg::bicgstab_solver<viennacl::vector<float> > solver(config);
	solver.set_initial_guess(x);
	x = solver(A, b);
	viennacl::copy(x, h_x);

	for (int pixel = 0; pixel < rows * cols; pixel++)
		depthImage[pixel] = h_x[pixel];
#else
	std::cout << "ViennaCL not supported" << std::endl;
#endif	
}
